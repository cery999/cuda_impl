#include "hip/hip_runtime.h"
// includes, system
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// includes, project
#include <hip/hip_runtime_api.h>  // helper functions for CUDA error checking and initialization
#include <helper_functions.h>  // helper utility functions

#ifdef __cplusplus
extern "C" {
#endif

#define BLAKE3_VERSION_STRING "1.3.3"
#define BLAKE3_KEY_LEN 32
#define BLAKE3_OUT_LEN 32
#define BLAKE3_BLOCK_LEN 64
#define BLAKE3_CHUNK_LEN 1024
#define BLAKE3_MAX_DEPTH 54

__constant__ uint32_t IV[8]{0x6A09E667UL, 0xBB67AE85UL, 0x3C6EF372UL,
                               0xA54FF53AUL, 0x510E527FUL, 0x9B05688CUL,
                               0x1F83D9ABUL, 0x5BE0CD19UL};


__global__ void blake3_compress_in_place_portable(uint32_t cv[8],
                                       const uint8_t block[BLAKE3_BLOCK_LEN],
                                       uint8_t block_len, uint64_t counter,
                                       uint8_t flags) {

}

void hash_one_portable(const uint8_t *input, size_t blocks,
                              const uint32_t key[8], uint64_t counter,
                              uint8_t flags, uint8_t flags_start,
                              uint8_t flags_end, uint8_t out[BLAKE3_OUT_LEN]) {
  uint32_t cv[8];
  memcpy(cv, key, BLAKE3_KEY_LEN);
  uint8_t block_flags = flags | flags_start;
  while (blocks > 0) {
    if (blocks == 1) {
      block_flags |= flags_end;
    }
    blake3_compress_in_place_portable(cv, input, BLAKE3_BLOCK_LEN, counter,
                                      block_flags);
    input = &input[BLAKE3_BLOCK_LEN];
    blocks -= 1;
    block_flags = flags;
  }
  store_cv_words(out, cv);
}

#ifdef __cplusplus
}
#endif


