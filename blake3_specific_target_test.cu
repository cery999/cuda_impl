#include "hip/hip_runtime.h"
// includes, system
#include <climits>
#include <cstdint>
#include <cstdlib>
#include <limits>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// CUDA runtime
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h> // helper functions for CUDA error checking and initialization
#include <helper_functions.h> // helper utility functions

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
using namespace cooperative_groups;

#define BLAKE3_VERSION_STRING "8.8.8"
#define BLAKE3_KEY_LEN 32
#define BLAKE3_OUT_LEN 32
#define BLAKE3_BLOCK_LEN 64
#define BLAKE3_CHUNK_LEN 1024
#define BLAKE3_MAX_DEPTH 54

#define INPUT_LEN 180
#define PARALLEL_DEGREE 1024000

// internal flags
enum blake3_flags {
  CHUNK_START = 1 << 0,
  CHUNK_END = 1 << 1,
  PARENT = 1 << 2,
  ROOT = 1 << 3,
};

static uint8_t *pined_inp[8], *pined_target[8];
static uint32_t *pined_out[8];
static uint64_t *pined_randoms[8];
static bool *pined_found[8];
static hipEvent_t event_start[8], event_stop[8];

__device__ __inline__ uint64_t to_big_end(uint64_t x) {
  // 1 2 3 4 5 6 7 8  -> 8 7 6 5 4 3 2 1
  return ((uint64_t)__byte_perm((uint32_t)(x), (uint32_t)(x >> 32), 0x0123)
          << 32) |
         __byte_perm((uint32_t)(x), (uint32_t)(x >> 32), 0x4567);
}

__global__ void to_big_end_launch(uint64_t *a, uint64_t *out) {
  *out = to_big_end(*a);
}

extern "C" void to_big_kernel() {
  void *a, *out;
  hipMalloc(&a, sizeof(uint64_t));
  hipMalloc(&out, sizeof(uint64_t));
  uint64_t l = 0x0102030405060708;
  uint8_t *ptr_l = (uint8_t *)&l;
  for (int i = 0; i < 8; i++) {
    printf("%02x", ptr_l[i]);
  }
  printf("\n");
  hipMemcpy(a, &l, sizeof(uint64_t), hipMemcpyHostToDevice);
  to_big_end_launch<<<1, 1>>>((uint64_t *)a, (uint64_t *)out);
  uint64_t oo;
  hipMemcpy(&oo, out, sizeof(uint64_t), hipMemcpyDeviceToHost);
  uint8_t *ptr = (uint8_t *)&oo;
  for (int i = 0; i < 8; i++) {
    printf("%02x", ptr[i]);
  }
  printf("\n");
  hipFree(a);
  hipFree(out);
}
__device__ uint32_t rotr32(uint32_t w, uint32_t c) {
  return (w >> c) | (w << (32 - c));
}
__device__ __inline__ uint32_t to_bigend(uint32_t x) {
  return __byte_perm(x, x, 0x0123);
}

#define PRINT_RESULT                                                           \
  do {                                                                         \
    for (auto i = 0; i < 32; i++) {                                            \
      printf("%02x", d_target[i]);                                             \
    }                                                                          \
    printf("\n");                                                              \
    for (auto i = 0; i < 4; i++) {                                             \
      printf("%02x", ((uint8_t *)&S0)[i]);                                     \
    }                                                                          \
    for (auto i = 0; i < 4; i++) {                                             \
      printf("%02x", ((uint8_t *)&S1)[i]);                                     \
    }                                                                          \
    for (auto i = 0; i < 4; i++) {                                             \
      printf("%02x", ((uint8_t *)&S2)[i]);                                     \
    }                                                                          \
    for (auto i = 0; i < 4; i++) {                                             \
      printf("%02x", ((uint8_t *)&S3)[i]);                                     \
    }                                                                          \
    for (auto i = 0; i < 4; i++) {                                             \
      printf("%02x", ((uint8_t *)&S4)[i]);                                     \
    }                                                                          \
    for (auto i = 0; i < 4; i++) {                                             \
      printf("%02x", ((uint8_t *)&S5)[i]);                                     \
    }                                                                          \
    for (auto i = 0; i < 4; i++) {                                             \
      printf("%02x", ((uint8_t *)&S6)[i]);                                     \
    }                                                                          \
    for (auto i = 0; i < 4; i++) {                                             \
      printf("%02x", ((uint8_t *)&S7)[i]);                                     \
    }                                                                          \
    printf("\n");                                                              \
  } while (0);

#define UPDATE_WITH_CV                                                         \
  do {                                                                         \
    S0 = 0x6A09E667UL;                                                         \
    S1 = 0xBB67AE85UL;                                                         \
    S2 = 0x3C6EF372UL;                                                         \
    S3 = 0xA54FF53AUL;                                                         \
    S4 = 0x510E527FUL;                                                         \
    S5 = 0x9B05688CUL;                                                         \
    S6 = 0x1F83D9ABUL;                                                         \
    S7 = 0x5BE0CD19UL;                                                         \
  } while (0);

#define UPDATE_WITH_CACHE                                                      \
  do {                                                                         \
    S0 = S0 ^ S8;                                                              \
    S1 = S1 ^ S9;                                                              \
    S2 = S2 ^ SA;                                                              \
    S3 = S3 ^ SB;                                                              \
    S4 = S4 ^ SC;                                                              \
    S5 = S5 ^ SD;                                                              \
    S6 = S6 ^ SE;                                                              \
    S7 = S7 ^ SF;                                                              \
  } while (0);

#define INIT(buf_len, flag)                                                    \
  do {                                                                         \
    S8 = 0x6A09E667UL;                                                         \
    S9 = 0xBB67AE85UL;                                                         \
    SA = 0x3C6EF372UL;                                                         \
    SB = 0xA54FF53AUL;                                                         \
    SC = 0;                                                                    \
    SD = 0;                                                                    \
    SE = (uint32_t)buf_len;                                                    \
    SF = (uint32_t)flag;                                                       \
  } while (0);

#define G(a, b, c, d, x, y)                                                    \
  do {                                                                         \
    a = a + b + x;                                                             \
    d = __byte_perm(d ^ a, d ^ a, 0x1032);                                     \
    c = c + d;                                                                 \
    b = rotr32(b ^ c, 12);                                                     \
    a = a + b + y;                                                             \
    d = __byte_perm(d ^ a, d ^ a, 0x0321);                                     \
    c = c + d;                                                                 \
    b = rotr32(b ^ c, 7);                                                      \
  } while (0);

#define UPDATE                                                                 \
  do {                                                                         \
    CV[0] = S0 ^ S8;                                                           \
    CV[1] = S1 ^ S9;                                                           \
    CV[2] = S2 ^ SA;                                                           \
    CV[3] = S3 ^ SB;                                                           \
    CV[4] = S4 ^ SC;                                                           \
    CV[5] = S5 ^ SD;                                                           \
    CV[6] = S6 ^ SE;                                                           \
    CV[7] = S7 ^ SF;                                                           \
  } while (0);

#define ROUND                                                                  \
  do {                                                                         \
    G(S0, S4, S8, SC, M[0], M[1]);                                             \
    G(S1, S5, S9, SD, M[2], M[3]);                                             \
    G(S2, S6, SA, SE, M[4], M[5]);                                             \
    G(S3, S7, SB, SF, M[6], M[7]);                                             \
    G(S0, S5, SA, SF, M[8], M[9]);                                             \
    G(S1, S6, SB, SC, M[10], M[11]);                                           \
    G(S2, S7, S8, SD, M[12], M[13]);                                           \
    G(S3, S4, S9, SE, M[14], M[15]);                                           \
    G(S0, S4, S8, SC, M[2], M[6]);                                             \
    G(S1, S5, S9, SD, M[3], M[10]);                                            \
    G(S2, S6, SA, SE, M[7], M[0]);                                             \
    G(S3, S7, SB, SF, M[4], M[13]);                                            \
    G(S0, S5, SA, SF, M[1], M[11]);                                            \
    G(S1, S6, SB, SC, M[12], M[5]);                                            \
    G(S2, S7, S8, SD, M[9], M[14]);                                            \
    G(S3, S4, S9, SE, M[15], M[8]);                                            \
    G(S0, S4, S8, SC, M[3], M[4]);                                             \
    G(S1, S5, S9, SD, M[10], M[12]);                                           \
    G(S2, S6, SA, SE, M[13], M[2]);                                            \
    G(S3, S7, SB, SF, M[7], M[14]);                                            \
    G(S0, S5, SA, SF, M[6], M[5]);                                             \
    G(S1, S6, SB, SC, M[9], M[0]);                                             \
    G(S2, S7, S8, SD, M[11], M[15]);                                           \
    G(S3, S4, S9, SE, M[8], M[1]);                                             \
    G(S0, S4, S8, SC, M[10], M[7]);                                            \
    G(S1, S5, S9, SD, M[12], M[9]);                                            \
    G(S2, S6, SA, SE, M[14], M[3]);                                            \
    G(S3, S7, SB, SF, M[13], M[15]);                                           \
    G(S0, S5, SA, SF, M[4], M[0]);                                             \
    G(S1, S6, SB, SC, M[11], M[2]);                                            \
    G(S2, S7, S8, SD, M[5], M[8]);                                             \
    G(S3, S4, S9, SE, M[1], M[6]);                                             \
    G(S0, S4, S8, SC, M[12], M[13]);                                           \
    G(S1, S5, S9, SD, M[9], M[11]);                                            \
    G(S2, S6, SA, SE, M[15], M[10]);                                           \
    G(S3, S7, SB, SF, M[14], M[8]);                                            \
    G(S0, S5, SA, SF, M[7], M[2]);                                             \
    G(S1, S6, SB, SC, M[5], M[3]);                                             \
    G(S2, S7, S8, SD, M[0], M[1]);                                             \
    G(S3, S4, S9, SE, M[6], M[4]);                                             \
    G(S0, S4, S8, SC, M[9], M[14]);                                            \
    G(S1, S5, S9, SD, M[11], M[5]);                                            \
    G(S2, S6, SA, SE, M[8], M[12]);                                            \
    G(S3, S7, SB, SF, M[15], M[1]);                                            \
    G(S0, S5, SA, SF, M[13], M[3]);                                            \
    G(S1, S6, SB, SC, M[0], M[10]);                                            \
    G(S2, S7, S8, SD, M[2], M[6]);                                             \
    G(S3, S4, S9, SE, M[4], M[7]);                                             \
    G(S0, S4, S8, SC, M[11], M[15]);                                           \
    G(S1, S5, S9, SD, M[5], M[0]);                                             \
    G(S2, S6, SA, SE, M[1], M[9]);                                             \
    G(S3, S7, SB, SF, M[8], M[6]);                                             \
    G(S0, S5, SA, SF, M[14], M[10]);                                           \
    G(S1, S6, SB, SC, M[2], M[12]);                                            \
    G(S2, S7, S8, SD, M[3], M[4]);                                             \
    G(S3, S4, S9, SE, M[7], M[13]);                                            \
  } while (0);

__global__ void special_launch(uint8_t *d_header, uint64_t start, uint64_t end,
                               size_t stride, uint8_t *d_target, uint32_t *out,
                               uint64_t *block_random_idx, bool *block_found) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t random_i = start + idx * stride; // parallel random message with i
  auto found = false;
  auto cta = this_thread_block();
  auto tile = tiled_partition<32>(cta);
  auto grid = this_grid();
  __shared__ bool thread_tile_group_found[17];
  __shared__ uint64_t thread_tile_group_random[17];
  if (random_i <= end) {
    // init chunk state
    // buf_len = 0, blocks_compressed = 0, flag = 0;
    uint32_t M[16] = {0}; // message blocks
    uint32_t S0, S1, S2, S3, S4, S5, S6, S7, S8, S9, SA, SB, SC, SD, SE,
        SF; // the state var

    // process first block with 64B with 180 - 64 remain
    uint32_t h_random_i = random_i >> 32, low_random_i = (uint32_t)(random_i);
    M[0] = __byte_perm(h_random_i, h_random_i, 0x0123);
    M[1] = __byte_perm(low_random_i, low_random_i, 0x0123);
    for (auto i = 0; i < 3; i++) {
      *(reinterpret_cast<int4 *>(&M[i * 4 + 2])) =
          *(reinterpret_cast<int4 *>(&d_header[i * 16]));
    }
    *(reinterpret_cast<int2 *>(&M[14])) =
        *(reinterpret_cast<int2 *>(&d_header[48]));

    /* printf("message: "); */
    /* for (auto i = 0; i < 64; i++) { */
    /*   printf("%02x", ((uint8_t *)M)[i]); */
    /* } */
    /* printf("\n"); */

    // init states
    UPDATE_WITH_CV;
    INIT(BLAKE3_BLOCK_LEN, CHUNK_START);
    // round 0 - 6
    ROUND;
    // update chain value in place
    /* UPDAET; */

    /* printf("%d:%d,%d:%d,%d:%d,%d:%d,%d:%d,%d:%d,%d:%d,%d:%d,%d:%d,%d:%d,%d:%d,%"
     */
    /*        "d:%d,%d:%d,%d:%d,%d:%d,%d:%d,\n", */
    /*        0, S0, 1, S1, 2, S2, 3, S3, 4, S4, 5, S5, 6, S6, 7, S7, 8, S8, 9,
     * S9, */
    /*        10, SA, 11, SB, 12, SC, 13, SD, 14, SE, 15, SF); */

    // blocks_compressed = 1 remain 116
#pragma unroll
    for (auto i = 0; i < 4; i++) {
      *(reinterpret_cast<int4 *>(&M[i * 4])) =
          *(reinterpret_cast<int4 *>(&d_header[i * 16 + 56]));
    }
    /* d_header += BLAKE3_BLOCK_LEN; */

    // init states
    UPDATE_WITH_CACHE
    INIT(BLAKE3_BLOCK_LEN, 0);
    // round 0 - 6
    ROUND;
    // update chain value in place
    /* UPDATE; */
    // blocks_compressed = 2 remain 52 do final

#pragma unroll
    for (auto i = 0; i < 3; i++) {
      *(reinterpret_cast<int4 *>(&M[i * 4])) =
          *(reinterpret_cast<int4 *>(&d_header[i * 16 + 56 + 64]));
    }
    *(reinterpret_cast<int2 *>(&M[13])) = make_int2(0, 0);
    M[15] = 0;
    d_header += 52; // remain 0

    // init states
    UPDATE_WITH_CACHE
    INIT(52, CHUNK_END | ROOT);
    // round 0 - 6
    ROUND;
    // done output will be chain value

    // for debug
    uint32_t *self_out = out + idx * 8;
    self_out[0] = S0 ^ S8;
    self_out[1] = S1 ^ S9;
    self_out[2] = S2 ^ SA;
    self_out[3] = S3 ^ SB;
    self_out[4] = S4 ^ SC;
    self_out[5] = S5 ^ SD;
    self_out[6] = S6 ^ SE;
    self_out[7] = S7 ^ SF;

    S0 ^= S8;
    S1 ^= S9;
    S2 ^= SA;
    S3 ^= SB;
    S4 ^= SC;
    S5 ^= SD;
    S6 ^= SE;
    S7 ^= SF;

    uint32_t CV[8];
    *reinterpret_cast<uint4 *>(&CV[0]) = make_uint4(S0, S1, S2, S3);
    *reinterpret_cast<uint4 *>(&CV[4]) = make_uint4(S4, S5, S6, S7);

    /* if (random_i <= end) { */
    /*   for (int i = 0; i < 32; i++) { */
    /*     printf("%02x", ((uint8_t *)CV)[i]); */
    /*   } */
    /*   printf("\n"); */
    /* } */

    auto is_break = false;
    for (auto i = 0; i < 32; i++) {
      if (((uint8_t *)CV)[i] < d_target[i]) {
        is_break = true;
        found = true;
        break;
      }
      if (((uint8_t *)CV)[i] > d_target[i]) {
        is_break = true;
        found = false;
        break;
      }
    }
    if (!is_break) {
      found = true; // equal
    }
    /* printf("compare_hash found: %d , random : %d\n", found, random_i); */
    /* if (idx == 0) { */
    /*   PRINT_RESULT; */
    /* } */
  }

  bool warp_found = false;
  uint64_t warp_random_idx = found ? random_i : UINT64_MAX;

  warp_found = tile.any(found);
  warp_random_idx = reduce(tile, warp_random_idx, less<uint64_t>());

  if (tile.thread_rank() == 0) {
    thread_tile_group_found[tile.meta_group_rank()] = warp_found;
    thread_tile_group_random[tile.meta_group_rank()] = warp_random_idx;
    /* printf("warp_rank: %d: found %d, random: %lld, thredid: %lld\n", */
    /*        tile.meta_group_rank(), warp_found, warp_random_idx, */
    /*        cta.thread_rank()); */
  }
  sync(cta);

  if (tile.meta_group_rank() == 0) {
    bool warp_group_found = false;
    uint64_t warp_group_random = UINT64_MAX;
    if (tile.thread_rank() < 16) {
      warp_group_found = thread_tile_group_found[tile.thread_rank()];
      warp_group_random = thread_tile_group_random[tile.thread_rank()];
    }

    for (auto offset = 8; offset > 0; offset >>= 1) {
      warp_group_found |=
          __shfl_down_sync(0x000000ff, warp_group_found, offset);
      warp_group_random =
          min(__shfl_down_sync(0x000000ff, warp_group_random, offset),
              warp_group_random);
    }
    if (tile.thread_rank() == 0 && tile.meta_group_rank() == 0) {
      block_found[grid.block_rank()] = warp_group_found;
      block_random_idx[grid.block_rank()] = warp_group_random;
      /* printf("blockid: %d: found %d, random: %lld, thredid: %lld\n",
       * blockIdx.x, */
      /*        warp_group_found, warp_group_random, cta.thread_rank()); */
    }
  }
}

__global__ void reduceGlobalBlocks(bool *global_found, uint64_t *global_random,
                                   uint64_t num) {
  volatile __shared__ bool shared_found[32];
  volatile __shared__ uint64_t shared_random[32];
  auto block = this_thread_block();
  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  unsigned int tid = threadIdx.x;
  unsigned int gridSize = block.size() * gridDim.x;
  unsigned int maskLength = (block.size() & 31); // 31 = warpSize-1
  maskLength = (maskLength > 0) ? (32 - maskLength) : maskLength;
  const unsigned int mask = (0xffffffff) >> maskLength;

  bool found = false;
  uint64_t random = UINT64_MAX;

  if ((block.size() & (block.size() - 1)) == 0) {
    unsigned int i = blockIdx.x * block.size() * 2 + threadIdx.x;
    gridSize = gridSize << 1;
    while (i < num) {
      found |= global_found[i];
      random = min(global_random[i], random);
      // ensure we don't read out of bounds -- this is optimized away for
      // powerOf2 sized arrays
      if ((i + block.size()) < num) {
        found |= global_found[i + block.size()];
        random = min(global_random[i + block.size()], random);
      }
      /* printf("global1: %d: found %d, random: %lld, thredid: %lld\n", i,
       * found, */
      /*        random, threadIdx.x); */

      i += gridSize;
    }
  } else {
    unsigned int i = blockIdx.x * block.size() + threadIdx.x;
    while (i < num) {
      found |= global_found[i];
      random = min(random, global_random[i]);

      /* printf("global2: %d: found %d, random: %lld, thredid: %lld\n", i,
       * found, */
      /*        random, threadIdx.x); */
      i += gridSize;
    }
  }

  for (int offset = warpSize / 2; offset > 0; offset /= 2) {
    auto warp_found = found;
    auto warp_random = random;
    warp_found |= __shfl_down_sync(mask, warp_found, offset);
    warp_random = min(__shfl_down_sync(mask, warp_random, offset), warp_random);
    if (threadIdx.x + offset < block.size()) {
      found = warp_found;
      random = warp_random;
    }
    /* printf("found: %d, random: %lld\n", found, random); */
  }

  if ((tid % warpSize) == 0) {
    shared_found[tid / warpSize] = found;
    shared_random[tid / warpSize] = random;
  }
  __syncthreads();

  const unsigned int shmem_extent =
      block.size() / warpSize > 0 ? block.size() / warpSize : 1;
  /* printf("shared mem extend %d\n", shmem_extent); */
  const unsigned int ballot_result = __ballot_sync(mask, tid < shmem_extent);
  if (tid < shmem_extent) {
    found = shared_found[tid];
    random = shared_random[tid];
    // Reduce final warp using shuffle or reduce_add if T==int & CUDA_ARCH ==
    // SM 8.0
    auto wrap_id = tid / warpSize;
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
      auto warp_group_found = found;
      auto warp_group_random = random;
      warp_group_found |=
          __shfl_down_sync(ballot_result, warp_group_found, offset);
      warp_group_random =
          min(warp_group_random,
              __shfl_down_sync(ballot_result, warp_group_random, offset));
      if ((tid + offset) < shmem_extent) {
        found = warp_group_found;
        random = warp_group_random;
        /* printf("reduce warp found: %d random: %lld, curid: %d , extent:
         * %d\n", */
        /*        found, random, tid + offset, shmem_extent); */
      }
    }
  }

  // write result for this block to global mem
  if (tid == 0) {
    global_found[blockIdx.x] = found;
    global_random[blockIdx.x] = random;

    /* printf("return global: %d: found %d, random: %lld, thredid: %d\n", */
    /*        blockIdx.x, found, random, threadIdx.x); */
  }
}

#ifdef __cplusplus
extern "C" {
#endif

unsigned int nextPow2(unsigned int x) {
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}

extern "C" void special_cuda_target(const uint8_t *header, uint64_t start,
                                    uint64_t end, size_t stride,
                                    const uint8_t target[32],
                                    uint64_t *host_randoms, uint32_t *found,
                                    uint8_t device_id) {
  hipProfilerStart();
  size_t num = ceil((end - start) * 1.0 / stride);
  dim3 block;
  dim3 grid;
  block = dim3(512, 1, 1);
  grid = dim3(ceil(num * 1.0 / 512), 1, 1);
  hipEventRecord(event_start[device_id], 0);
  hipMemcpyAsync(pined_inp[device_id], header + 8, INPUT_LEN - 8,
                  hipMemcpyHostToDevice, 0);
  hipMemcpyAsync(pined_target[device_id], target, BLAKE3_OUT_LEN,
                  hipMemcpyHostToDevice);
  printf("launch %d gride %d block\n", grid.x, block.x);
  special_launch<<<grid, block>>>(
      pined_inp[device_id], start, end, stride, pined_target[device_id],
      pined_out[device_id], pined_randoms[device_id], pined_found[device_id]);
  checkCudaErrors(hipGetLastError());
  uint8_t *host_out = (uint8_t *)malloc(32 * 1024);
  hipMemcpyAsync(host_out, pined_out[device_id], 32 * 1024,
                  hipMemcpyDeviceToHost);
  // find blocks found and random
  auto total_block_num = grid.x;
  if (total_block_num >= 1024) {
    block = dim3(1024, 1, 1);
  } else {
    block = dim3(total_block_num, 1, 1);
  }
  grid = dim3(ceil((total_block_num * 1.0) / 1024), 1, 1);
  if (block.x > 1) {
    printf("launch %d grid %d block reduce\n", grid.x, block.x);
    reduceGlobalBlocks<<<grid, block>>>(pined_found[device_id],
                                        pined_randoms[device_id], grid.x);
  }
  checkCudaErrors(hipGetLastError());
  bool global_found;
  hipMemcpyAsync(&global_found, pined_found[device_id], sizeof(bool),
                  hipMemcpyDeviceToHost);
  hipMemcpyAsync(host_randoms, pined_randoms[device_id], sizeof(uint64_t),
                  hipMemcpyDeviceToHost);
  hipEventRecord(event_stop[device_id], 0);
  hipEventSynchronize(event_stop[device_id]);
  *found = (uint32_t)global_found;
  hipProfilerStop();

  printf("found hash: \n");

  for (auto i = 0; i < 1; i++) {
    for (size_t i = 0; i < BLAKE3_OUT_LEN; i++) {
      printf("%02x", host_out[i]);
    }
    printf("\n");
    host_out += 32;
  }
}

extern "C" void pre_allocate(uint8_t device_id) {
  printf("allocate device %d\n", device_id);
  hipSetDevice(device_id);
  hipEventCreate(&event_start[device_id]);
  hipEventCreate(&event_stop[device_id]);
  hipMalloc((void **)&pined_inp[device_id], INPUT_LEN);
  hipMalloc((void **)&pined_out[device_id], 32 * PARALLEL_DEGREE);
  hipMalloc((void **)&pined_target[device_id], 32);
  hipMalloc((void **)&pined_found[device_id], sizeof(bool) * 102400);
  hipMalloc((void **)&pined_randoms[device_id], sizeof(uint64_t) * 102400);
}

extern "C" void post_free(uint8_t device_id) {
  hipEventDestroy(event_start[device_id]);
  hipEventDestroy(event_stop[device_id]);
  hipFree(pined_inp[device_id]);
  hipFree(pined_target[device_id]);
}

extern "C" void getDeviceNum(int32_t *nums) {
  hipGetDeviceCount(nums);
  printf("detect %d nums gpu\n", *nums);
}

#ifdef __cplusplus
}
#endif
