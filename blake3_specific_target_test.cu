#include "hip/hip_runtime.h"
// includes, system
#include <cstdint>
#include <cstdlib>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// CUDA runtime
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h> // helper functions for CUDA error checking and initialization
#include <helper_functions.h> // helper utility functions

#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;

#ifdef __cplusplus
extern "C" {
#endif

#define BLAKE3_VERSION_STRING "8.8.8"
#define BLAKE3_KEY_LEN 32
#define BLAKE3_OUT_LEN 32
#define BLAKE3_BLOCK_LEN 64
#define BLAKE3_CHUNK_LEN 1024
#define BLAKE3_MAX_DEPTH 54

#define INPUT_LEN 180
#define PARALLEL_DEGREE 1024000

// internal flags
enum blake3_flags {
  CHUNK_START = 1 << 0,
  CHUNK_END = 1 << 1,
  PARENT = 1 << 2,
  ROOT = 1 << 3,
};

static uint8_t *pined_inp[8], *pined_target[8];
static uint32_t *pined_out[8];
static uint64_t *pined_randoms[8];
static bool *pined_found[8];
static hipEvent_t event_start[8], event_stop[8];

__device__ __inline__ uint64_t to_big_end(uint64_t x) {
  // 1 2 3 4 5 6 7 8  -> 8 7 6 5 4 3 2 1
  return ((uint64_t)__byte_perm((uint32_t)(x), (uint32_t)(x >> 32), 0x0123)
          << 32) |
         __byte_perm((uint32_t)(x), (uint32_t)(x >> 32), 0x4567);
}

__global__ void to_big_end_launch(uint64_t *a, uint64_t *out) {
  *out = to_big_end(*a);
}

extern "C" void to_big_kernel() {
  void *a, *out;
  hipMalloc(&a, sizeof(uint64_t));
  hipMalloc(&out, sizeof(uint64_t));
  uint64_t l = 0x0102030405060708;
  uint8_t *ptr_l = (uint8_t *)&l;
  for (int i = 0; i < 8; i++) {
    printf("%02x", ptr_l[i]);
  }
  printf("\n");
  hipMemcpy(a, &l, sizeof(uint64_t), hipMemcpyHostToDevice);
  to_big_end_launch<<<1, 1>>>((uint64_t *)a, (uint64_t *)out);
  uint64_t oo;
  hipMemcpy(&oo, out, sizeof(uint64_t), hipMemcpyDeviceToHost);
  uint8_t *ptr = (uint8_t *)&oo;
  for (int i = 0; i < 8; i++) {
    printf("%02x", ptr[i]);
  }
  printf("\n");
  hipFree(a);
  hipFree(out);
}
__device__ uint32_t rotr32(uint32_t w, uint32_t c) {
  return (w >> c) | (w << (32 - c));
}
__device__ __inline__ uint32_t to_bigend(uint32_t x) {
  return __byte_perm(x, x, 0x0123);
}

#define PRINT_RESULT                                                           \
  do {                                                                         \
    for (auto i = 0; i < 32; i++) {                                            \
      printf("%02x", d_target[i]);                                             \
    }                                                                          \
    printf("\n");                                                              \
    for (auto i = 0; i < 4; i++) {                                             \
      printf("%02x", ((uint8_t *)&S0)[i]);                                     \
    }                                                                          \
    for (auto i = 0; i < 4; i++) {                                             \
      printf("%02x", ((uint8_t *)&S1)[i]);                                     \
    }                                                                          \
    for (auto i = 0; i < 4; i++) {                                             \
      printf("%02x", ((uint8_t *)&S2)[i]);                                     \
    }                                                                          \
    for (auto i = 0; i < 4; i++) {                                             \
      printf("%02x", ((uint8_t *)&S3)[i]);                                     \
    }                                                                          \
    for (auto i = 0; i < 4; i++) {                                             \
      printf("%02x", ((uint8_t *)&S4)[i]);                                     \
    }                                                                          \
    for (auto i = 0; i < 4; i++) {                                             \
      printf("%02x", ((uint8_t *)&S5)[i]);                                     \
    }                                                                          \
    for (auto i = 0; i < 4; i++) {                                             \
      printf("%02x", ((uint8_t *)&S6)[i]);                                     \
    }                                                                          \
    for (auto i = 0; i < 4; i++) {                                             \
      printf("%02x", ((uint8_t *)&S7)[i]);                                     \
    }                                                                          \
    printf("\n");                                                              \
  } while (0);

#define UPDATE_WITH_CV                                                         \
  do {                                                                         \
    S0 = 0x6A09E667UL;                                                         \
    S1 = 0xBB67AE85UL;                                                         \
    S2 = 0x3C6EF372UL;                                                         \
    S3 = 0xA54FF53AUL;                                                         \
    S4 = 0x510E527FUL;                                                         \
    S5 = 0x9B05688CUL;                                                         \
    S6 = 0x1F83D9ABUL;                                                         \
    S7 = 0x5BE0CD19UL;                                                         \
  } while (0);

#define UPDATE_WITH_CACHE                                                      \
  do {                                                                         \
    S0 = S0 ^ S8;                                                              \
    S1 = S1 ^ S9;                                                              \
    S2 = S2 ^ SA;                                                              \
    S3 = S3 ^ SB;                                                              \
    S4 = S4 ^ SC;                                                              \
    S5 = S5 ^ SD;                                                              \
    S6 = S6 ^ SE;                                                              \
    S7 = S7 ^ SF;                                                              \
  } while (0);

#define INIT(buf_len, flag)                                                    \
  do {                                                                         \
    S8 = 0x6A09E667UL;                                                         \
    S9 = 0xBB67AE85UL;                                                         \
    SA = 0x3C6EF372UL;                                                         \
    SB = 0xA54FF53AUL;                                                         \
    SC = 0;                                                                    \
    SD = 0;                                                                    \
    SE = (uint32_t)buf_len;                                                    \
    SF = (uint32_t)flag;                                                       \
  } while (0);

#define G(a, b, c, d, x, y)                                                    \
  do {                                                                         \
    a = a + b + x;                                                             \
    d = __byte_perm(d ^ a, d ^ a, 0x1032);                                     \
    c = c + d;                                                                 \
    b = rotr32(b ^ c, 12);                                                     \
    a = a + b + y;                                                             \
    d = __byte_perm(d ^ a, d ^ a, 0x0321);                                     \
    c = c + d;                                                                 \
    b = rotr32(b ^ c, 7);                                                      \
  } while (0);

#define UPDATE                                                                 \
  do {                                                                         \
    CV[0] = S0 ^ S8;                                                           \
    CV[1] = S1 ^ S9;                                                           \
    CV[2] = S2 ^ SA;                                                           \
    CV[3] = S3 ^ SB;                                                           \
    CV[4] = S4 ^ SC;                                                           \
    CV[5] = S5 ^ SD;                                                           \
    CV[6] = S6 ^ SE;                                                           \
    CV[7] = S7 ^ SF;                                                           \
  } while (0);

#define ROUND                                                                  \
  do {                                                                         \
    G(S0, S4, S8, SC, M[0], M[1]);                                             \
    G(S1, S5, S9, SD, M[2], M[3]);                                             \
    G(S2, S6, SA, SE, M[4], M[5]);                                             \
    G(S3, S7, SB, SF, M[6], M[7]);                                             \
    G(S0, S5, SA, SF, M[8], M[9]);                                             \
    G(S1, S6, SB, SC, M[10], M[11]);                                           \
    G(S2, S7, S8, SD, M[12], M[13]);                                           \
    G(S3, S4, S9, SE, M[14], M[15]);                                           \
    G(S0, S4, S8, SC, M[2], M[6]);                                             \
    G(S1, S5, S9, SD, M[3], M[10]);                                            \
    G(S2, S6, SA, SE, M[7], M[0]);                                             \
    G(S3, S7, SB, SF, M[4], M[13]);                                            \
    G(S0, S5, SA, SF, M[1], M[11]);                                            \
    G(S1, S6, SB, SC, M[12], M[5]);                                            \
    G(S2, S7, S8, SD, M[9], M[14]);                                            \
    G(S3, S4, S9, SE, M[15], M[8]);                                            \
    G(S0, S4, S8, SC, M[3], M[4]);                                             \
    G(S1, S5, S9, SD, M[10], M[12]);                                           \
    G(S2, S6, SA, SE, M[13], M[2]);                                            \
    G(S3, S7, SB, SF, M[7], M[14]);                                            \
    G(S0, S5, SA, SF, M[6], M[5]);                                             \
    G(S1, S6, SB, SC, M[9], M[0]);                                             \
    G(S2, S7, S8, SD, M[11], M[15]);                                           \
    G(S3, S4, S9, SE, M[8], M[1]);                                             \
    G(S0, S4, S8, SC, M[10], M[7]);                                            \
    G(S1, S5, S9, SD, M[12], M[9]);                                            \
    G(S2, S6, SA, SE, M[14], M[3]);                                            \
    G(S3, S7, SB, SF, M[13], M[15]);                                           \
    G(S0, S5, SA, SF, M[4], M[0]);                                             \
    G(S1, S6, SB, SC, M[11], M[2]);                                            \
    G(S2, S7, S8, SD, M[5], M[8]);                                             \
    G(S3, S4, S9, SE, M[1], M[6]);                                             \
    G(S0, S4, S8, SC, M[12], M[13]);                                           \
    G(S1, S5, S9, SD, M[9], M[11]);                                            \
    G(S2, S6, SA, SE, M[15], M[10]);                                           \
    G(S3, S7, SB, SF, M[14], M[8]);                                            \
    G(S0, S5, SA, SF, M[7], M[2]);                                             \
    G(S1, S6, SB, SC, M[5], M[3]);                                             \
    G(S2, S7, S8, SD, M[0], M[1]);                                             \
    G(S3, S4, S9, SE, M[6], M[4]);                                             \
    G(S0, S4, S8, SC, M[9], M[14]);                                            \
    G(S1, S5, S9, SD, M[11], M[5]);                                            \
    G(S2, S6, SA, SE, M[8], M[12]);                                            \
    G(S3, S7, SB, SF, M[15], M[1]);                                            \
    G(S0, S5, SA, SF, M[13], M[3]);                                            \
    G(S1, S6, SB, SC, M[0], M[10]);                                            \
    G(S2, S7, S8, SD, M[2], M[6]);                                             \
    G(S3, S4, S9, SE, M[4], M[7]);                                             \
    G(S0, S4, S8, SC, M[11], M[15]);                                           \
    G(S1, S5, S9, SD, M[5], M[0]);                                             \
    G(S2, S6, SA, SE, M[1], M[9]);                                             \
    G(S3, S7, SB, SF, M[8], M[6]);                                             \
    G(S0, S5, SA, SF, M[14], M[10]);                                           \
    G(S1, S6, SB, SC, M[2], M[12]);                                            \
    G(S2, S7, S8, SD, M[3], M[4]);                                             \
    G(S3, S4, S9, SE, M[7], M[13]);                                            \
  } while (0);

#define CHECK_TARGET(N, P)                                                     \
  do {                                                                         \
    if ((((S##N ^ S##P) >> 0) & 0xff) > (*(d_target + N * 4 + 0)))             \
      return;                                                                  \
    if ((((S##N ^ S##P) >> 8) & 0xff) > (*(d_target + N * 4 + 1)))             \
      return;                                                                  \
    if ((((S##N ^ S##P) >> 16) & 0xff) > (*(d_target + N * 4 + 2)))            \
      return;                                                                  \
    if ((((S##N ^ S##P) >> 24) & 0xff) > (*(d_target + N * 4 + 3)))            \
      return;                                                                  \
  } while (0);

#define CHECK_I(i)                                                             \
  do {                                                                         \
    if (S##i > d_target_u32[i]) {                                              \
      found = false;                                                           \
      goto reduce;                                                             \
    } else if (S##i < d_target_u32[i]) {                                       \
      found = true;                                                            \
      goto reduce;                                                             \
    }                                                                          \
  } while (0);

__global__ void special_launch(uint8_t *d_header, uint64_t start, uint64_t end,
                               size_t stride, uint8_t *d_target, uint32_t *out,
                               uint64_t *block_random_idx, bool *block_found) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t random_i = start + idx * stride; // parallel random message with i
  auto found = false;
  thread_block cta = this_thread_block();
  __shared__ uint64_t shared_found[32];
  if (random_i < end) {
    // init chunk state
    // buf_len = 0, blocks_compressed = 0, flag = 0;
    uint32_t M[16] = {0}; // message blocks
    uint32_t S0, S1, S2, S3, S4, S5, S6, S7, S8, S9, SA, SB, SC, SD, SE,
        SF; // the state var

    // process first block with 64B with 180 - 64 remain
    uint32_t h_random_i = random_i >> 32, low_random_i = (uint32_t)(random_i);
    M[0] = __byte_perm(h_random_i, h_random_i, 0x0123);
    M[1] = __byte_perm(low_random_i, low_random_i, 0x0123);
    for (auto i = 0; i < 3; i++) {
      *(reinterpret_cast<int4 *>(&M[i * 4 + 2])) =
          *(reinterpret_cast<int4 *>(&d_header[i * 16]));
    }
    *(reinterpret_cast<int2 *>(&M[14])) =
        *(reinterpret_cast<int2 *>(&d_header[48]));

    /* printf("message: "); */
    /* for (auto i = 0; i < 64; i++) { */
    /*   printf("%02x", ((uint8_t *)M)[i]); */
    /* } */
    /* printf("\n"); */
    /* d_header += 56; */

    // init states
    UPDATE_WITH_CV;
    INIT(BLAKE3_BLOCK_LEN, CHUNK_START);
    // round 0 - 6
    ROUND;
    // update chain value in place
    /* UPDAET; */

    /* printf("%d:%d,%d:%d,%d:%d,%d:%d,%d:%d,%d:%d,%d:%d,%d:%d,%d:%d,%d:%d,%d:%d,%"
     */
    /*        "d:%d,%d:%d,%d:%d,%d:%d,%d:%d,\n", */
    /*        0, S0, 1, S1, 2, S2, 3, S3, 4, S4, 5, S5, 6, S6, 7, S7, 8, S8, 9,
     * S9, */
    /*        10, SA, 11, SB, 12, SC, 13, SD, 14, SE, 15, SF); */

    // blocks_compressed = 1 remain 116
#pragma unroll
    for (auto i = 0; i < 4; i++) {
      *(reinterpret_cast<int4 *>(&M[i * 4])) =
          *(reinterpret_cast<int4 *>(&d_header[i * 16 + 56]));
    }
    /* d_header += BLAKE3_BLOCK_LEN; */

    // init states
    UPDATE_WITH_CACHE
    INIT(BLAKE3_BLOCK_LEN, 0);
    // round 0 - 6
    ROUND;
    // update chain value in place
    /* UPDATE; */
    // blocks_compressed = 2 remain 52 do final

#pragma unroll
    for (auto i = 0; i < 3; i++) {
      *(reinterpret_cast<int4 *>(&M[i * 4])) =
          *(reinterpret_cast<int4 *>(&d_header[i * 16 + 56 + 64]));
    }
    *(reinterpret_cast<int2 *>(&M[13])) = make_int2(0, 0);
    M[15] = 0;
    d_header += 52; // remain 0

    // init states
    UPDATE_WITH_CACHE
    INIT(52, CHUNK_END | ROOT);
    // round 0 - 6
    ROUND;
    // done output will be chain value

    // for debug
    uint32_t *self_out = out + idx * 8;
    self_out[0] = S0 ^ S8;
    self_out[1] = S1 ^ S9;
    self_out[2] = S2 ^ SA;
    self_out[3] = S3 ^ SB;
    self_out[4] = S4 ^ SC;
    self_out[5] = S5 ^ SD;
    self_out[6] = S6 ^ SE;
    self_out[7] = S7 ^ SF;

    S0 ^= S8;
    S1 ^= S9;
    S2 ^= SA;
    S3 ^= SB;
    S4 ^= SC;
    S5 ^= SD;
    S6 ^= SE;
    S7 ^= SF;

    uint32_t *d_target_u32 = (uint32_t *)d_target;
    if (idx == 0) {
      PRINT_RESULT;
    }

    CHECK_I(0);
    CHECK_I(1);
    CHECK_I(2);
    CHECK_I(3);
    CHECK_I(4);
    CHECK_I(5);
    CHECK_I(6);
    CHECK_I(7);
    found = true; // equal
                  // do block reduce and save to global
  reduce:;

    thread_block_tile<32> tile = tiled_partition<32>(cta);
    tile.sync();
    bool warp_found = false;
    uint64_t warp_random_idx = random_i;
    warp_found = tile.any(found);
    if (warp_found) {
      for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        warp_random_idx = __shfl_down_sync(0xffffffff, warp_random_idx, offset);
        warp_random_idx = min(random_i, warp_random_idx);
      }
    }
    __shared__ bool thread_tile_group_found[16];
    __shared__ uint64_t thread_tile_group_random[16];
    if (tile.thread_rank() == 0) {
      thread_tile_group_found[tile.meta_group_rank()] = found;
      thread_tile_group_random[tile.meta_group_rank()] = warp_random_idx;
    }
    sync(cta);

    bool warp_group_found = false;
    uint64_t warp_group_random = warp_random_idx;
    if (tile.meta_group_rank() == 0) {
      if (tile.thread_rank() < tile.meta_group_size()) {
        warp_group_found =
            __any_sync(0x0000ffff, thread_tile_group_found[tile.thread_rank()]);

        for (int offset = tile.meta_group_size() / 2; offset > 0; offset /= 2) {
          warp_group_random =
              __shfl_down_sync(0x0000ffff, warp_group_random, offset);
          warp_group_random = min(warp_random_idx, warp_group_random);
        }
      }
      tile.sync();
      if (tile.thread_rank() == 0) {
        block_found[cta.group_index().x] = warp_group_found;
        block_random_idx[cta.group_index().x] = warp_group_random;
      }
    }
  }
}

__global__ void global_block_reduce(bool *global_found, uint64_t *global_random,
                                    uint64_t blockSize) {
  auto cta = this_thread_block();
  auto warp = tiled_partition<32>(cta);
  auto idx = cta.thread_rank();
  __shared__ bool shared_global_found[32];
  if (idx < blockSize) {
    auto block_found = global_found[idx];
    auto warp_found = warp.any(block_found);
    uint64_t warp_random = global_random[idx];
    uint64_t self_random = warp_random;
    for (auto offset = warpSize / 2; offset > 0; offset /= 2) {
      self_random = min(warp.shfl_down(self_random, offset), self_random);
    }
    if (warp.thread_rank() == 0) {
      shared_global_found[warp.meta_group_rank()] = warp_found;
    }
    sync(cta);

    if (warp.meta_group_rank() == 0) {
      auto global_shared_found =
          warp.any(shared_global_found[warp.thread_rank()]);
      if (global_shared_found && warp.thread_rank() == 0) {
        global_found[0] = global_shared_found;
      }
    }
  }
}

extern "C" void special_cuda_target(const uint8_t *header, uint64_t start,
                                    uint64_t end, size_t stride,
                                    const uint8_t target[32],
                                    uint64_t *host_randoms, uint32_t *found,
                                    uint8_t device_id) {
  hipProfilerStart();
  size_t num = (end - start) / stride;
  dim3 block;
  dim3 grid;
  if (num > 1024) {
    block = dim3(1024, 1, 1);
  } else {
    block = dim3(num, 1, 1);
  }
  grid = dim3(ceil(num * 1.0 / 1024), 1, 1);
  hipEventRecord(event_start[device_id], 0);
  hipMemcpyAsync(pined_inp[device_id], header + 8, INPUT_LEN - 8,
                  hipMemcpyHostToDevice, 0);
  hipMemcpyAsync(pined_target[device_id], target, BLAKE3_OUT_LEN,
                  hipMemcpyHostToDevice);
  hipMemsetAsync(pined_found[device_id], 0, 1024 * sizeof(bool));
  hipMemsetAsync(pined_randoms[device_id], 0, 1024 * sizeof(uint64_t));
#define BLOCK_SIZE 32
  special_launch<<<BLOCK_SIZE, 512>>>(
      pined_inp[device_id], start, end, stride, pined_target[device_id],
      pined_out[device_id], pined_randoms[device_id], pined_found[device_id]);
  checkCudaErrors(hipGetLastError());

  hipMemcpyAsync(found, pined_found[device_id], sizeof(bool),
                  hipMemcpyDeviceToHost);
  hipMemcpyAsync(host_randoms, pined_randoms[device_id], sizeof(uint64_t),
                  hipMemcpyDeviceToHost);
  uint8_t *host_out = (uint8_t *)malloc(32 * 1024);
  hipMemcpyAsync(host_out, pined_out[device_id], 32 * 1024,
                  hipMemcpyDeviceToHost);
  hipEventRecord(event_stop[device_id], 0);
  hipEventSynchronize(event_stop[device_id]);
  hipProfilerStop();

  for (auto i = 0; i < 1; i++) {
    for (size_t i = 0; i < BLAKE3_OUT_LEN; i++) {
      printf("%02x", host_out[i]);
    }
    printf("\n");
    host_out += 32;
  }
}

extern "C" void pre_allocate(uint8_t device_id) {
  printf("allocate device %d\n", device_id);
  hipSetDevice(device_id);
  hipEventCreate(&event_start[device_id]);
  hipEventCreate(&event_stop[device_id]);
  hipMalloc((void **)&pined_inp[device_id], INPUT_LEN);
  hipMalloc((void **)&pined_out[device_id], 32 * PARALLEL_DEGREE);
  hipMalloc((void **)&pined_target[device_id], 32);
  hipMalloc((void **)&pined_found[device_id], sizeof(bool) * 1024);
  hipMalloc((void **)&pined_randoms[device_id], sizeof(uint64_t) * 1024);
}

extern "C" void post_free(uint8_t device_id) {
  hipEventDestroy(event_start[device_id]);
  hipEventDestroy(event_stop[device_id]);
  hipFree(pined_inp[device_id]);
  hipFree(pined_target[device_id]);
}

extern "C" void getDeviceNum(int32_t *nums) {
  hipGetDeviceCount(nums);
  printf("detect %d nums gpu\n", *nums);
}

#ifdef __cplusplus
}
#endif
