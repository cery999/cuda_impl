#include "hip/hip_runtime.h"
// includes, system
#include <cstdint>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// CUDA runtime
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h> // helper functions for CUDA error checking and initialization
#include <helper_functions.h> // helper utility functions

#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;

#ifdef __cplusplus
extern "C" {
#endif

#define BLAKE3_VERSION_STRING "8.8.8"
#define BLAKE3_KEY_LEN 32
#define BLAKE3_OUT_LEN 32
#define BLAKE3_BLOCK_LEN 64
#define BLAKE3_CHUNK_LEN 1024
#define BLAKE3_MAX_DEPTH 54

#define INPUT_LEN 180
#define PARALLEL_DEGREE 1024000

// internal flags
enum blake3_flags {
  CHUNK_START = 1 << 0,
  CHUNK_END = 1 << 1,
  PARENT = 1 << 2,
  ROOT = 1 << 3,
};

static uint8_t *pined_inp[8], *pined_target[8];
static uint32_t *pined_out[8];
static uint64_t *pined_randoms[8];
static bool *pined_found[8];
static hipEvent_t event_start[8], event_stop[8];

__device__ __inline__ uint64_t to_big_end(uint64_t x) {
  // 1 2 3 4 5 6 7 8  -> 8 7 6 5 4 3 2 1
  return ((uint64_t)__byte_perm((uint32_t)(x), (uint32_t)(x >> 32), 0x0123)
          << 32) |
         __byte_perm((uint32_t)(x), (uint32_t)(x >> 32), 0x4567);
}

__global__ void to_big_end_launch(uint64_t *a, uint64_t *out) {
  *out = to_big_end(*a);
}

extern "C" void to_big_kernel() {
  void *a, *out;
  hipMalloc(&a, sizeof(uint64_t));
  hipMalloc(&out, sizeof(uint64_t));
  uint64_t l = 0x0102030405060708;
  uint8_t *ptr_l = (uint8_t *)&l;
  for (int i = 0; i < 8; i++) {
    printf("%02x", ptr_l[i]);
  }
  printf("\n");
  hipMemcpy(a, &l, sizeof(uint64_t), hipMemcpyHostToDevice);
  to_big_end_launch<<<1, 1>>>((uint64_t *)a, (uint64_t *)out);
  uint64_t oo;
  hipMemcpy(&oo, out, sizeof(uint64_t), hipMemcpyDeviceToHost);
  uint8_t *ptr = (uint8_t *)&oo;
  for (int i = 0; i < 8; i++) {
    printf("%02x", ptr[i]);
  }
  printf("\n");
  hipFree(a);
  hipFree(out);
}
__device__ uint32_t rotr32(uint32_t w, uint32_t c) {
  return (w >> c) | (w << (32 - c));
}
__device__ __inline__ uint32_t to_bigend(uint32_t x) {
  return __byte_perm(x, x, 0x0123);
}

#define UPDATE_WITH_CV                                                         \
  do {                                                                         \
    S0 = 0x6A09E667UL;                                                         \
    S1 = 0xBB67AE85UL;                                                         \
    S2 = 0x3C6EF372UL;                                                         \
    S3 = 0xA54FF53AUL;                                                         \
    S4 = 0x510E527FUL;                                                         \
    S5 = 0x9B05688CUL;                                                         \
    S6 = 0x1F83D9ABUL;                                                         \
    S7 = 0x5BE0CD19UL;                                                         \
  } while (0);

#define UPDATE_WITH_CACHE                                                      \
  do {                                                                         \
    S0 = S0 ^ S8;                                                              \
    S1 = S1 ^ S9;                                                              \
    S2 = S2 ^ SA;                                                              \
    S3 = S3 ^ SB;                                                              \
    S4 = S4 ^ SC;                                                              \
    S5 = S5 ^ SD;                                                              \
    S6 = S6 ^ SE;                                                              \
    S7 = S7 ^ SF;                                                              \
  } while (0);

#define INIT(buf_len, flag)                                                    \
  do {                                                                         \
    S8 = 0x6A09E667UL;                                                         \
    S9 = 0xBB67AE85UL;                                                         \
    SA = 0x3C6EF372UL;                                                         \
    SB = 0xA54FF53AUL;                                                         \
    SC = 0;                                                                    \
    SD = 0;                                                                    \
    SE = (uint32_t)buf_len;                                                    \
    SF = (uint32_t)flag;                                                       \
  } while (0);

#define G(a, b, c, d, x, y)                                                    \
  do {                                                                         \
    a = a + b + x;                                                             \
    d = rotr32(d ^ a, 16);                                                     \
    c = c + d;                                                                 \
    b = rotr32(b ^ c, 12);                                                     \
    a = a + b + y;                                                             \
    d = rotr32(d ^ a, 8);                                                      \
    c = c + d;                                                                 \
    b = rotr32(b ^ c, 7);                                                      \
  } while (0);

#define UPDATE                                                                 \
  do {                                                                         \
    CV[0] = S0 ^ S8;                                                           \
    CV[1] = S1 ^ S9;                                                           \
    CV[2] = S2 ^ SA;                                                           \
    CV[3] = S3 ^ SB;                                                           \
    CV[4] = S4 ^ SC;                                                           \
    CV[5] = S5 ^ SD;                                                           \
    CV[6] = S6 ^ SE;                                                           \
    CV[7] = S7 ^ SF;                                                           \
  } while (0);

#define ROUND                                                                  \
  do {                                                                         \
    G(S0, S4, S8, SC, M[0], M[1]);                                             \
    G(S1, S5, S9, SD, M[2], M[3]);                                             \
    G(S2, S6, SA, SE, M[4], M[5]);                                             \
    G(S3, S7, SB, SF, M[6], M[7]);                                             \
    G(S0, S5, SA, SF, M[8], M[9]);                                             \
    G(S1, S6, SB, SC, M[10], M[11]);                                           \
    G(S2, S7, S8, SD, M[12], M[13]);                                           \
    G(S3, S4, S9, SE, M[14], M[15]);                                           \
    G(S0, S4, S8, SC, M[2], M[6]);                                             \
    G(S1, S5, S9, SD, M[3], M[10]);                                            \
    G(S2, S6, SA, SE, M[7], M[0]);                                             \
    G(S3, S7, SB, SF, M[4], M[13]);                                            \
    G(S0, S5, SA, SF, M[1], M[11]);                                            \
    G(S1, S6, SB, SC, M[12], M[5]);                                            \
    G(S2, S7, S8, SD, M[9], M[14]);                                            \
    G(S3, S4, S9, SE, M[15], M[8]);                                            \
    G(S0, S4, S8, SC, M[3], M[4]);                                             \
    G(S1, S5, S9, SD, M[10], M[12]);                                           \
    G(S2, S6, SA, SE, M[13], M[2]);                                            \
    G(S3, S7, SB, SF, M[7], M[14]);                                            \
    G(S0, S5, SA, SF, M[6], M[5]);                                             \
    G(S1, S6, SB, SC, M[9], M[0]);                                             \
    G(S2, S7, S8, SD, M[11], M[15]);                                           \
    G(S3, S4, S9, SE, M[8], M[1]);                                             \
    G(S0, S4, S8, SC, M[10], M[7]);                                            \
    G(S1, S5, S9, SD, M[12], M[9]);                                            \
    G(S2, S6, SA, SE, M[14], M[3]);                                            \
    G(S3, S7, SB, SF, M[13], M[15]);                                           \
    G(S0, S5, SA, SF, M[4], M[0]);                                             \
    G(S1, S6, SB, SC, M[11], M[2]);                                            \
    G(S2, S7, S8, SD, M[5], M[8]);                                             \
    G(S3, S4, S9, SE, M[1], M[6]);                                             \
    G(S0, S4, S8, SC, M[12], M[13]);                                           \
    G(S1, S5, S9, SD, M[9], M[11]);                                            \
    G(S2, S6, SA, SE, M[15], M[10]);                                           \
    G(S3, S7, SB, SF, M[14], M[8]);                                            \
    G(S0, S5, SA, SF, M[7], M[2]);                                             \
    G(S1, S6, SB, SC, M[5], M[3]);                                             \
    G(S2, S7, S8, SD, M[0], M[1]);                                             \
    G(S3, S4, S9, SE, M[6], M[4]);                                             \
    G(S0, S4, S8, SC, M[9], M[14]);                                            \
    G(S1, S5, S9, SD, M[11], M[5]);                                            \
    G(S2, S6, SA, SE, M[8], M[12]);                                            \
    G(S3, S7, SB, SF, M[15], M[1]);                                            \
    G(S0, S5, SA, SF, M[13], M[3]);                                            \
    G(S1, S6, SB, SC, M[0], M[10]);                                            \
    G(S2, S7, S8, SD, M[2], M[6]);                                             \
    G(S3, S4, S9, SE, M[4], M[7]);                                             \
    G(S0, S4, S8, SC, M[11], M[15]);                                           \
    G(S1, S5, S9, SD, M[5], M[0]);                                             \
    G(S2, S6, SA, SE, M[1], M[9]);                                             \
    G(S3, S7, SB, SF, M[8], M[6]);                                             \
    G(S0, S5, SA, SF, M[14], M[10]);                                           \
    G(S1, S6, SB, SC, M[2], M[12]);                                            \
    G(S2, S7, S8, SD, M[3], M[4]);                                             \
    G(S3, S4, S9, SE, M[7], M[13]);                                            \
  } while (0);

#define CHECK_TARGET(N, P)                                                     \
  do {                                                                         \
    if ((((S##N ^ S##P) >> 0) & 0xff) > (*(d_target + N * 4 + 0)))             \
      return;                                                                  \
    if ((((S##N ^ S##P) >> 8) & 0xff) > (*(d_target + N * 4 + 1)))             \
      return;                                                                  \
    if ((((S##N ^ S##P) >> 16) & 0xff) > (*(d_target + N * 4 + 2)))            \
      return;                                                                  \
    if ((((S##N ^ S##P) >> 24) & 0xff) > (*(d_target + N * 4 + 3)))            \
      return;                                                                  \
  } while (0);

__global__ void special_launch(uint8_t *d_header, uint64_t start, uint64_t end,
                               size_t stride, uint8_t *d_target, uint32_t *out,
                               uint64_t *random_idx, bool *found) {
  auto idx = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t random_i = start + idx * stride; // parallel random message with i

  if (random_i < end) {
    // init chunk state
    // buf_len = 0, blocks_compressed = 0, flag = 0;
    uint32_t CV[8];
    uint32_t M[16] = {0}; // message blocks
    uint32_t S0, S1, S2, S3, S4, S5, S6, S7, S8, S9, SA, SB, SC, SD, SE,
        SF; // the state var

    // process first block with 64B with 180 - 64 remain
    uint32_t h_random_i = random_i >> 32, low_random_i = (uint32_t)(random_i);
    M[0] = __byte_perm(h_random_i, h_random_i, 0x0123);
    M[1] = __byte_perm(low_random_i, low_random_i, 0x0123);
    for (auto i = 0; i < 14; i++) {
      M[i + 2] = *((uint32_t *)d_header + i);
    }
    /* printf("message: "); */
    /* for (auto i = 0; i < 64; i++) { */
    /*   printf("%02x", ((uint8_t *)M)[i]); */
    /* } */
    /* printf("\n"); */
    d_header += 56;

    // init states
    UPDATE_WITH_CV;
    INIT(BLAKE3_BLOCK_LEN, CHUNK_START);
    // round 0 - 6
    ROUND;
    // update chain value in place
    /* UPDAET; */

    /* printf("%d:%d,%d:%d,%d:%d,%d:%d,%d:%d,%d:%d,%d:%d,%d:%d,%d:%d,%d:%d,%d:%d,%"
     */
    /*        "d:%d,%d:%d,%d:%d,%d:%d,%d:%d,\n", */
    /*        0, S0, 1, S1, 2, S2, 3, S3, 4, S4, 5, S5, 6, S6, 7, S7, 8, S8, 9,
     * S9, */
    /*        10, SA, 11, SB, 12, SC, 13, SD, 14, SE, 15, SF); */

    // blocks_compressed = 1 remain 116
#pragma unroll
    for (auto i = 0; i < 16; i++) {
      M[i] = *((uint32_t *)d_header + i);
    }
    d_header += BLAKE3_BLOCK_LEN;

    // init states
    UPDATE_WITH_CACHE
    INIT(BLAKE3_BLOCK_LEN, 0);
    // round 0 - 6
    ROUND;
    // update chain value in place
    /* UPDATE; */
    // blocks_compressed = 2 remain 52 do final

#pragma unroll
    for (auto i = 0; i < 13; i++) {
      M[i] = *((uint32_t *)d_header + i);
    }

#pragma unroll
    for (auto i = 13; i < 16; i++) {
      M[i] = 0;
    }
    d_header += 52; // remain 0

    // init states
    UPDATE_WITH_CACHE
    INIT(52, CHUNK_END | ROOT);
    // round 0 - 6
    ROUND;
    UPDATE;
    // done output will be chain value

    // for debug
    uint32_t *self_out = out + idx * 8;
    self_out[0] = S0 ^ S8;
    self_out[1] = S1 ^ S9;
    self_out[2] = S2 ^ SA;
    self_out[3] = S3 ^ SB;
    self_out[4] = S4 ^ SC;
    self_out[5] = S5 ^ SD;
    self_out[6] = S6 ^ SE;
    self_out[7] = S7 ^ SF;

#pragma unroll
    for (auto i = 0; i < 32; i++) {
      if (((uint8_t *)&CV)[i] > ((uint8_t *)&d_target)[i])
        return;
      if (((uint8_t *)&CV)[i] < d_target[i]) {
        *found = true;
        *random_idx = (uint64_t)atomicMin((unsigned long long int *)random_idx,
                                          (unsigned long long int)random_i);
        return;
      }
    }

    // match i
    *found = true;
    // may be fault on mac,x32 system
    *random_idx = (uint64_t)atomicMin((unsigned long long int *)random_idx,
                                      (unsigned long long int)random_i);
  }
}

extern "C" void special_cuda_target(const uint8_t *header, uint64_t start,
                                    uint64_t end, size_t stride,
                                    const uint8_t target[32],
                                    uint64_t *host_randoms, bool *found,
                                    uint8_t device_id) {
  hipProfilerStart();
  size_t num = (end - start) / stride;
  dim3 block;
  dim3 grid;
  if (num > 1024) {
    block = dim3(1024, 1, 1);
  } else {
    block = dim3(num, 1, 1);
  }
  grid = dim3(ceil(num * 1.0 / 1024), 1, 1);
  hipEventRecord(event_start[device_id], 0);
  hipMemcpyAsync(pined_inp[device_id], header + 8, INPUT_LEN - 8,
                  hipMemcpyHostToDevice, 0);
  hipMemcpyAsync(pined_target[device_id], target, BLAKE3_OUT_LEN,
                  hipMemcpyHostToDevice);
  hipMemsetAsync(pined_found[device_id], 0, sizeof(bool));
  special_launch<<<1, 1024>>>(pined_inp[device_id], start, end, stride,
                              pined_target[device_id], pined_out[device_id],
                              pined_randoms[device_id], pined_found[device_id]);
  checkCudaErrors(hipGetLastError());
  hipMemcpyAsync(found, pined_found[device_id], sizeof(bool),
                  hipMemcpyDeviceToHost);
  hipMemcpyAsync(host_randoms, pined_randoms[device_id], sizeof(uint64_t),
                  hipMemcpyDeviceToHost);
  uint8_t *host_out = (uint8_t *)malloc(32 * 1024);
  hipMemcpyAsync(host_out, pined_out[device_id], 32 * 1024,
                  hipMemcpyDeviceToHost);
  hipEventRecord(event_stop[device_id], 0);
  hipProfilerStop();

  for (auto i = 0; i < 1; i++) {
    for (size_t i = 0; i < BLAKE3_OUT_LEN; i++) {
      printf("%02x", host_out[i]);
    }
    printf("\n");
    host_out += 32;
  }
}

extern "C" void pre_allocate(uint8_t device_id) {
  printf("allocate device %d\n", device_id);
  hipSetDevice(device_id);
  hipEventCreate(&event_start[device_id]);
  hipEventCreate(&event_stop[device_id]);
  hipMalloc((void **)&pined_inp[device_id], INPUT_LEN);
  hipMalloc((void **)&pined_out[device_id], 32 * PARALLEL_DEGREE);
  hipMalloc((void **)&pined_target[device_id], 32);
  hipMalloc(&pined_found[device_id], sizeof(bool));
  hipMalloc(&pined_randoms[device_id], sizeof(uint64_t) * 2);
}

extern "C" void post_free(uint8_t device_id) {
  hipEventDestroy(event_start[device_id]);
  hipEventDestroy(event_stop[device_id]);
  hipFree(pined_inp[device_id]);
  hipFree(pined_target[device_id]);
}

extern "C" void getDeviceNum(int32_t *nums) {
  hipGetDeviceCount(nums);
  printf("detect %d nums gpu\n", *nums);
}

#ifdef __cplusplus
}
#endif
